#include "hip/hip_runtime.h"
#include "src/kernels/transpose_kernel.h"
#include <iostream>
//if MQA or GQA, we should use this transpose to broadcast kv head num to q head num
//[num layers, bs, kv head num, max_seq_len, head size]=>[bs, q head num, max_k_len, head size]
//context_length.shape=[bs]
// 这个kernel叫repeat_interleave或者broadcast比较合理
template<typename T>
__global__ void transpose_value_cache(T*          v_dst, 
                                      const T*    v_src,
                                      const size_t layer_offset,
                                      const int    head_num,
                                      const int    q_head_per_kv,
                                      const int    head_size,
                                      const int*   context_length,
                                      const int    max_k_len,
                                      const int    max_seq_len)
{
    const int batch_id = blockIdx.y;
    const int head_id  = blockIdx.z;

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    const auto val_src = v_src + layer_offset;
    const auto val_dst = v_dst;

    const auto seq_len = context_length[batch_id];

    const int v_head_size_id = idx % head_size;
    const int v_seq_len_id   = idx / head_size;
    // only fetch context_length(<max_seq_len) kv data from all kv cache of current seq
    if (v_seq_len_id < seq_len) {
        const int64_t src_idx = batch_id * (head_num / q_head_per_kv) * head_size * max_seq_len + //B
                                head_id / q_head_per_kv * head_size * max_seq_len +  // H
                                v_seq_len_id * head_size +                        // s
                                v_head_size_id;                                             // D/x

        const int64_t dst_idx = batch_id * head_num * head_size * max_k_len +  // B
                                head_id * head_size * max_k_len +              // H
                                v_seq_len_id * head_size +                      // s
                                v_head_size_id;                                           // D/x

        val_dst[dst_idx] = val_src[src_idx];
    }
}
template<typename T>
void launchTransposeKVCache(TensorWrapper<T>* k_cache_src,
                            TensorWrapper<T>* v_cache_src,
                            TensorWrapper<int>* context_length,
                            TensorWrapper<int>* layer_id,
                            TensorWrapper<T>* k_cache_dst,
                            TensorWrapper<T>* v_cache_dst
                            )
{
    int batch_size = context_length->shape[0];
    int kv_head_num = k_cache_src->shape[1];
    int max_seq_len = k_cache_src->shape[2];
    int head_num = k_cache_dst->shape[1];
    
    int max_k_len = k_cache_dst->shape[2];
    int head_size = k_cache_dst->shape[3];
    int layer = layer_id->getVal();
    //note: here MUSTN'T use layer_id->getVal<int>(), because we cant access GPU memory directly by [] if data is on GPU
    //note: so we can make layer data locate on CPU
    size_t layer_offset = layer * batch_size * kv_head_num * max_seq_len * head_size;
    int q_head_per_kv = head_num / kv_head_num;
    int blockSize = 128;
    dim3 block(128);
    dim3 grid((max_k_len * head_size + blockSize - 1) / blockSize, batch_size, head_num); // q head num
    std::cout << "calling transpose/broadcast kernel" << "\n";    
    transpose_value_cache<T><<<grid, block>>>(v_cache_dst->data, 
                                              v_cache_src->data,
                                              layer_offset,
                                              head_num,
                                              q_head_per_kv,
                                              head_size,
                                              context_length->data,
                                              max_k_len,
                                              max_seq_len);
                                              
    transpose_value_cache<T><<<grid, block>>>(k_cache_dst->data, 
                                              k_cache_src->data,
                                              layer_offset,
                                              head_num,
                                              q_head_per_kv,
                                              head_size,
                                              context_length->data,
                                              max_k_len,
                                              max_seq_len);
    std::cout << "called transpose/broadcast kernel" << "\n";

}

template void launchTransposeKVCache(TensorWrapper<float>* k_cache_src,
                            TensorWrapper<float>* v_cache_src,
                            TensorWrapper<int>* context_length,
                            TensorWrapper<int>* layer_id,
                            TensorWrapper<float>* k_cache_dst,
                            TensorWrapper<float>* v_cache_dst
                            );
template void launchTransposeKVCache(TensorWrapper<half>* k_cache_src,
                            TensorWrapper<half>* v_cache_src,
                            TensorWrapper<int>* context_length,
                            TensorWrapper<int>* layer_id,
                            TensorWrapper<half>* k_cache_dst,
                            TensorWrapper<half>* v_cache_dst
                            );