#include "hip/hip_runtime.h"
#include "src/kernels/transpose_kernel.h"
#include <iostream>
//[bs, kv head num, max_seq_len, head size]=>[bs, q head num, max_k_len, head size]
//context_length.shape=[bs]
template<typename T>
__global__ void transpose_value_cache(T*          v_dst, 
                                      const T*    v_src,
                                      const size_t layer_offset,
                                      const int    head_num,
                                      const int    q_head_per_kv,
                                      const int    head_size,
                                      const int*   context_length,
                                      const int    max_k_len,
                                      const int    max_seq_len)
{
    const int batch_id = blockIdx.y;
    const int head_id  = blockIdx.z;

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    const auto val_src = v_src + layer_offset;
    const auto val_dst = v_dst;

    const auto seq_len = context_length[batch_id];

    const int v_head_size_id = idx % head_size;
    const int v_seq_len_id   = idx / head_size;
    // only fetch context_length(<max_seq_len) kv data from all kv cache of current seq
    if (v_seq_len_id < seq_len) {
        const int64_t src_idx = batch_id * (head_num / q_head_per_kv) * head_size * max_seq_len + //B
                                head_id / q_head_per_kv * head_size * max_seq_len +  // H
                                v_seq_len_id * head_size +                        // s
                                v_head_size_id;                                             // D/x

        const int64_t dst_idx = batch_id * head_num * head_size * max_k_len +  // B
                                head_id * head_size * max_k_len +              // H
                                v_seq_len_id * head_size +                      // s
                                v_head_size_id;                                           // D/x

        val_dst[dst_idx] = val_src[src_idx];
    }
}

template<typename T>
void launchTransposeKVCache(Tensor* k_cache_src,
                            Tensor* v_cache_src,
                            Tensor* context_length,
                            size_t  layer_offset,
                            Tensor* k_cache_dst,
                            Tensor* v_cache_dst
                            )
{
    int batch_size = context_length->shape[0];
    int kv_head_num = k_cache_src->shape[1];
    int max_seq_len = k_cache_src->shape[2];
    int head_num = k_cache_dst->shape[1];
    int max_k_len = k_cache_dst->shape[2];
    int head_size = k_cache_dst->shape[3];

    int q_head_per_kv = head_num / kv_head_num;
    int blockSize = 128;
    dim3 block(128);
    dim3 grid((max_k_len * head_size + blockSize - 1) / blockSize, batch_size, head_num); // q head num
    transpose_value_cache<T><<<grid, block>>>((T*)v_cache_dst, 
                                              (const T*)v_cache_src,
                                              layer_offset,
                                              head_num,
                                              q_head_per_kv,
                                              head_size,
                                              (int*)context_length->data,
                                              max_k_len,
                                              max_seq_len);
                                              
    transpose_value_cache<T><<<grid, block>>>((T*)k_cache_dst, 
                                              (const T*)k_cache_src,
                                              layer_offset,
                                              head_num,
                                              q_head_per_kv,
                                              head_size,
                                              (int*)context_length->data,
                                              max_k_len,
                                              max_seq_len);
}
