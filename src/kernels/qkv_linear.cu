#include <iostream>
#include "src/kernels/qkv_linear.h"
//TODO: when abstracted weight class, replace T with class
//weight * input
//weight shape = [hidden_units, hidden_units]
//input shape = [hidden_units, seqlen]
// void launchLinearGemm(const float* input,
//                   float* output, 
//                   const int input_2nd_dim, 
//                   const float* weight,
//                   const int hidden_units){}
template<typename T>
void launchLinearGemm(Tensor* input,
                      BaseWeight<T>& weight, 
                      Tensor* output,
                      bool trans_a = false,
                      bool trans_b = false) {
    //TODO: enhance the below 3 obj and setgemmconfig created only once in highest file like ft/bert_example.cc
    hipStream_t stream;
    hipblasHandle_t cublas_handle;
    hipblasLtHandle_t cublaslt_handle;
    std::cout << "creating stream" << "\n";
    //hipStreamCreate(&stream);
    // !!!remember to call hipblasCreate to create cublas handle!fxxk nvidia, that spent me 1 day to check
    hipblasCreate(&cublas_handle);
    hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
    std::cout << "creating cublaswrapper" << "\n";
    cublasWrapper* cublas_wrapper = 
                        new cublasWrapper(cublas_handle, cublaslt_handle);
                       // , stream);
    cublas_wrapper->setFP32GemmConfig();
    int input_lda = input->shape[0];
    int weight_ldb = input->shape[1];
    // TODO:check 2nd dim of input = 1st dim of weight
    int output_ldc = input_lda;         
    int k = output->shape[1];
    hipblasOperation_t transA = trans_a ? HIPBLAS_OP_T: HIPBLAS_OP_N;
    hipblasOperation_t transB = trans_b ? HIPBLAS_OP_T: HIPBLAS_OP_N;
    std::cout << "calling gemm" << "\n";
    cublas_wrapper->Gemm(transA,
                        transB,
                        input_lda,      //m
                        k,              //n
                        weight_ldb,     //k
                        (T*)input->data,   //A
                        input_lda,      //lda
                        (T*)weight.data,   //B
                        weight_ldb,     //ldb 
                        (T*)output->data,  //C
                        output_ldc,     //ldc   
                        1.0f,
                        0.0f);
}

// We must instancite the template, if not, will report linking issue
template void launchLinearGemm(Tensor* input, BaseWeight<float>& weight, Tensor* output, bool trans_a = false,
                                bool trans_b = false);

template<typename T>
void launchLinearStridedBatchGemm(Tensor* input1,
                                  Tensor* input2,
                                  Tensor* output,
                                  bool trans_a = false,
                                  bool trans_b = false)
{
    hipStream_t stream;
    hipblasHandle_t cublas_handle;
    hipblasLtHandle_t cublaslt_handle;
    std::cout << "creating stream" << "\n";
    hipblasCreate(&cublas_handle);
    hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
    std::cout << "creating cublaswrapper" << "\n";
    cublasWrapper* cublas_wrapper = 
                        new cublasWrapper(cublas_handle, cublaslt_handle);
                       // , stream);
    cublas_wrapper->setFP32GemmConfig();

    int m = input1->shape[2];
    int k = input1->shape[3];
    int n = input2->shape[2];
    int lda = m;
    int ldb = k;
    int ldc = m;
    int64_t strideA = m * k;
    int64_t strideB = k * n;
    int64_t strideC = m * n;
    // TODO:check 4nd dim of input = 3rd dim of weight
    // TODO:check batchCount of two matrix is equal
    int batchCount = input1->shape[0] * input1->shape[1];

    std::cout << "calling batch gemm" << "\n";
    hipblasOperation_t transA = trans_a ? HIPBLAS_OP_T: HIPBLAS_OP_N;
    hipblasOperation_t transB = trans_b ? HIPBLAS_OP_T: HIPBLAS_OP_N;
    cublas_wrapper->stridedBatchedGemm(transA,
                                       transB,
                                       m,
                                       n,
                                       k,
                                       (T*)input1->data, //A
                                       lda,
                                       strideA,
                                       (T*)input2->data, //B
                                       ldb,
                                       strideB,
                                       (T*)output->data, //C
                                       ldc,
                                       strideC,
                                       batchCount,
                                       1.0f,
                                       0.0f);
}
template void launchLinearStridedBatchGemm<float>(Tensor* input1, Tensor* input2, Tensor* output, bool trans_a = false,
                                        bool trans_b = false);
