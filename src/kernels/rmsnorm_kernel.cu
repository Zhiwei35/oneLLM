#include "hip/hip_runtime.h"
#include <stdio.h>
#include "src/kernels/rmsnorm_kernel.h"
//bugs1: 2nd warpreducesum returns 0, because blockDim.x < 32, blockDim.x / 32=0
//bugs2: output buffer valuse is the same as ones before call, thats because we didn't successfully write into the output address
//bugs3: output buffer's 1st 32 values are right, the latter is wrong, because when we use vec, the ele nums of a row is hiddenunits/vecsize, we should note the row stride to move the ptr carefully
template<typename T>
struct Vec {};

template<>
struct Vec<half> {
   using Type = half2;
   static constexpr int size = 2;
};

template<>
struct Vec<float> {
    using Type = float4;
    static constexpr int size = 4;
};
template<typename T>
__device__ T warpReduceSum(T val){
    for(int i = 32 / 2; i > 0; i >>= 1){
        val += __shfl_xor_sync(0xffffffff, val, i);
    }
    return val; // 32 threads return val, but only 0th thread is sum val
}
//note:!!!when blocksize < 32, use blockDim.x/32 to get warp nums is wrong, we should instead ceil it
template<typename T>
__device__ T blockReduceSum(T val){
    int tid = threadIdx.x;
    int wid = tid / 32;
    int laneid = tid % 32;
    int warpnum = (blockDim.x + 31) / 32;
    static __shared__ T warpsum[64];
    val = warpReduceSum<T>(val);
    if(laneid == 0){
        warpsum[wid] = val;
    }
    __syncthreads();

    T sum = tid < warpnum ? warpsum[tid] : 0;
    sum = warpReduceSum<T>(sum); //though 0th own the sum, but dont need to shfl sync
    return sum;
}
// 1.this kernel is used after self attention and FFN in every layer
// 2.I allocate threads number by assuming head size can be divided by 4 and 2
__global__ void RMSNorm(float* decoder_out, // [num tokens, q_hidden_units]
                        float* scale, //[q_hidden_units], RMSNorm weights
                        float eps, //RMSNorm eps
                        int num_tokens, 
                        int hidden_units){
    int vec_size = Vec<float>::size;
    using Vec_t = typename Vec<float>::Type;
    int batch_id = blockIdx.x;
    int tid = threadIdx.x;
    Vec_t* s;
    Vec_t dout, tmp;
    
    float thread_accm = static_cast<float>(0);

    for(int i = tid; i < hidden_units / vec_size; i += blockDim.x) {
        dout = reinterpret_cast<Vec_t*>(decoder_out)[batch_id * hidden_units / vec_size + i];// note the offset should divide vec size
        thread_accm += dout.x * dout.x + dout.y * dout.y + 
                       dout.z * dout.z + dout.w * dout.w;
    } //x^2
    
    // mean(x^2)
    float blocksum = blockReduceSum<float>(thread_accm);
    __shared__ float inv_fenmu;
    if(tid == 0){
        inv_fenmu = rsqrt(blocksum / hidden_units + eps);
    }
    // rmsnorm
    Vec_t* out = reinterpret_cast<Vec_t*>(decoder_out + batch_id * hidden_units);// note before vec the stride is batch_id * hiddenunits w/o / vecsize
    s = reinterpret_cast<Vec_t*>(const_cast<float*>(scale));
    for(int i = tid; i < hidden_units / vec_size; i += blockDim.x) {
        //s = reinterpret_cast<Vec_t*>(const_cast<T*>(scale))[i];
        out[i].x = s[i].x * out[i].x * inv_fenmu;
        out[i].y = s[i].y * out[i].y * inv_fenmu;
        out[i].z = s[i].z * out[i].z * inv_fenmu;
        out[i].w = s[i].w * out[i].w * inv_fenmu;
    }    
}

void launchRMSNorm( Tensor* decoder_out, // [num tokens, hidden_units]
                    LayerNormWeight& attn_norm_weight, //RMSNorm weights
                    float eps //RMSNorm eps
                    )
{
    int num_tokens = decoder_out->shape[0];
    int hidden_units = decoder_out->shape[1];
    int vec_size = Vec<float>::size;
    int num_threads = hidden_units / vec_size; // assume head size can be divided by 4 and 2
    dim3 grid(num_tokens);
    dim3 block(num_threads);
    printf("calling RMSNorm\n");
    RMSNorm<<<grid, block>>>((float*)decoder_out->data,
                            attn_norm_weight.gamma,
                            eps,
                            num_tokens,
                            hidden_units);
    printf("called RMSNorm\n");
}
