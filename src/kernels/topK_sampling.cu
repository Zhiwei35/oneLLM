#include "hip/hip_runtime.h"
#include <stdio.h>
#include "src/kernels/topK_sampling.h"
// mini-softmax + curand_sample
// input: [bs, K] from topK output
// output: [bs]
template<typename T>
__global__ void SamplingKernel(int* topk_id,
                               T* topk_val, //[bs,K] from topK
                               int* output_id, //[bs]
                               int* seqlen, //cu seqlen,[bs]
                               bool* is_finished, //[bs]
                               int K,
                               int rand_num, // that is step
                               int end_id, // when initialize llama model, we will init it, and this is a fixed val
                               int vocab_size)
{
    int batch_id = blockIdx.x;
    int bid = batch_id;
    int tid = threadIdx.x;
    int offset = batch_id * K + tid;
    if (tid == 0) {
        printf("topk_val max = %f\n", (float)topk_val[batch_id * K]);
        printf("topk_id[0] = %d\n", topk_id[0]);
    }

    float max_val = (float)(topk_val[batch_id * K]) ; // max val is the top of the buffer, because topK
    topk_val[offset] = expf((float)topk_val[offset] - max_val);

    __shared__ float thredhold, sum;
    if(tid == 0) {
        sum = 0.0f;
        for(int i = 0; i < K; i++) {
            sum += (float)topk_val[batch_id * K + i];
        }
        hiprandState_t state;
        hiprand_init((unsigned long long)rand_num,(unsigned long long)bid, (unsigned long long)0, &state);// not sure rand_num's type is suitable here or not
        thredhold = (float)hiprand_uniform(&state) * sum; // for a block
        printf("sampling info: \n");
        printf("sum = %f\n", sum);
        printf("thredhold = %f\n", thredhold);
        printf("top1 val = %f", (float)topk_val[0]);

        for(int i = 0; i < K; i++) {
            thredhold = thredhold - (float)topk_val[batch_id * K + i];
            if(thredhold < 0) {
                output_id[bid] = topk_id[batch_id * K + i] % vocab_size;
                break;
            }
        }
        seqlen[bid] = is_finished[bid] ? seqlen[bid] : seqlen[bid] + 1;
        is_finished[bid] = output_id[bid] == end_id ? 1 : 0;
    }
}
template<typename T>
void launchSampling(TensorWrapper<int>* topk_id,
                    TensorWrapper<T>* topk_val,
                    TensorWrapper<int>* seqlen,
                    TensorWrapper<bool>* is_finished,
                    TensorWrapper<int>* output_id,
                    TensorWrapper<int>* step,
                    IntDict& params) {
    int batch_size = topk_id->shape[0];
    int K = topk_id->shape[1];
    int vocab_size = params["vocab_size"];
    int step_ = step->getVal();
    int end_id = params["end_id"];

    dim3 grid(batch_size);
    dim3 block(K); // K is small, so directly allocate K threads is enough
    // std::cout << "calling sampling kernel" << "\n";
    SamplingKernel<T><<<grid, block>>>(
        topk_id->data,
        topk_val->data,
        output_id->data,
        seqlen->data,
        is_finished->data,
        K,
        step_,
        end_id,
        vocab_size
    );
    // std::cout << "called sampling kernel" << "\n";
                    }

template void launchSampling(TensorWrapper<int>* topk_id,
                    TensorWrapper<float>* topk_val,
                    TensorWrapper<int>* seqlen,
                    TensorWrapper<bool>* is_finished,
                    TensorWrapper<int>* output_id,
                    TensorWrapper<int>* step,
                    IntDict& params);
template void launchSampling(TensorWrapper<int>* topk_id,
                    TensorWrapper<half>* topk_val,
                    TensorWrapper<int>* seqlen,
                    TensorWrapper<bool>* is_finished,
                    TensorWrapper<int>* output_id,
                    TensorWrapper<int>* step,
                    IntDict& params);