#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include "src/kernels/decoder_masked_attn.h"

template<typename T>
__device__ T warpReduceSum(T val){

    for(int mask = 16; mask > 0; mask >>= 2){
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;

}
template<typename T>
__device__ T blockReduceSum(T val){
    int tid = threadIdx.x;
    int warp_id = tid / 32;
    int lane_id = tid % 32;
    int warp_nums = blockDim.x / 32;
    static __shared__ T warpsum[64];//why add static?or will report incomplete type
    // returned val is the sum computed by 0th thread.
    val = warpReduceSum<T>(val);
    //note: here return val of warpreducesum should be stored into smem , rather not reg, because here nums of return val are warp nums not thread nums.
    if (lane_id == 0){
        warpsum[warp_id] = val;
    }
    float warp_val = tid < warp_nums ? warpsum[warp_id] : 0;
    return warpReduceSum<T>(warp_val);

}
template<typename T>
__device__ T warpReduceMax(T val){

    for(int mask = 16; mask > 0; mask >>= 2){
        val = max(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}

template<typename T>
__device__ T blockReduceMax(T val){
    int tid = threadIdx.x;
    int warp_id = tid / 32;
    int lane_id = tid % 32;
    int warp_nums = blockDim.x / 32;
    static __shared__ T warpmax[64];
    // returned val is the max computed by 0th thread.
    val = warpReduceMax<T>(val);
    //note: here return val of warpreducemax should be stored into smem , rather not reg, because here nums of return val are warp nums not thread nums.
    if (lane_id == 0){
        warpmax[warp_id] = val;
    }
    float warp_val = tid < warp_nums ? warpmax[warp_id] : 0;
    return warpReduceMax<T>(warp_val);
}
// kv cache is the output of context attention(prompt phase), and the input of masked attention(token gen)
struct masked_MHA_kernel_params
{
    float* q;       //[bs, num heads, 1, head size]
    float* k;       //[bs, num heads, 1, head size]
    float* v;       //[bs, num heads, 1, head size]
    float* k_cache; //output,[max_seq_len or step, bs, num heads, head size] from prompt phase
    float* v_cache; //output,[max_seq_len or step, bs, num heads, head size] from prompt phase
    int batch_size;
    int num_heads;
    int head_size;
    int scale; // =rsqrt(head size);
    //TODO: add qkv bias
    int step;
    float* mha_output; //[bs, num heads, head size]
};
// block and thread allocation
// 1 block -> head size，后续可改进为1 warp -> 1 head size
// 1 grid -> bs * num heads
__global__ void masked_MHA_kernel(masked_MHA_kernel_params& params){
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int head_id = bid % params.num_heads;
    int batch_id = bid / params.num_heads;

    int batch_stride = params.num_heads * params.head_size;
    int head_stride = params.head_size;
    int qkv_offset = batch_id * batch_stride + head_id * head_stride + tid;
    int cache_offset = params.batch_size * batch_stride;

    const float* q_mem = params.q;
    const float* k_mem = params.k;
    const float* v_mem = params.v;

    // q k smem for block reduce
    extern __shared__ float sqk[];
    float* sq = sqk;
    float* sk = sq + params.head_size;
    float* logits = sk + params.head_size;
    float* sv = logits + params.step;
    // FT 2.1的写法里面，kv cache是在prompt阶段已经填充，iter=0为token gen的起始iter
    for(int iter = 0; iter < params.step; iter++) {
        // every iter,  q and k's shape = [1, head size]
        // reuse k cache
        // float k = params.k_cache[iter * cache_offset + qkv_offset];
        sk[tid]= params.k_cache[iter * cache_offset + qkv_offset];
        __syncthreads();
        // when final step, update k cache
        if (iter == params.step - 1) {
            // TODO: update k cache with k with bias add
            params.k_cache[iter * cache_offset + qkv_offset] = k_mem[qkv_offset];
            sk[tid] = k_mem[qkv_offset];
        }
        
        sq[tid] = q_mem[qkv_offset];
        __syncthreads();
        float qk = (tid < params.head_size) ? sq[tid] * sk[tid] * params.scale : (float)0.0f;
        //block reduce using multi warp reduce
        //TODO: maybe broadcast the attn score to each thread of the block in blockreducesum
        float attn_score = blockReduceSum<float>(qk);
        if(tid == 0) {
            logits[iter] = attn_score;
        }
        __syncthreads();
    }
    //softmax(logits), logits.shape = [bs, num heads, 1, step] 
    float local_logits = tid < params.step ? (float)logits[tid] : 0; 
    float row_max = blockReduceMax<float>(local_logits);
    float fenzi = tid < params.step ? expf(logits[tid] - row_max) : 0;
    float fenmu = blockReduceSum<float>(fenzi);
    if(tid < params.step) {
        logits[tid] = fenzi / fenmu;
    }
    __syncthreads();
    if(blockIdx.x==0 && tid==0) printf("in cuda kernel\n");
    // logits*V = [bs, num heads, 1, step] * [max_seq_len or step, bs, num heads, head size]
    if (tid < params.head_size) {  
        // note: here is head size ,not step, because step by step, we have to use [1, step/seqlen] from logits * [1, head size] from v
        // so here we use acc O to acc the one ele logits * one ele v every step iter
        float O = 0.0f;
        for(int iter = 0; iter < params.step; iter++) {
            sv[tid]= params.v_cache[iter * cache_offset + qkv_offset];
            __syncthreads();
            // when final step, update k cache
            if (iter == params.step - 1) {
                // TODO: update k cache with k with bias add
                params.v_cache[iter * cache_offset + qkv_offset] = v_mem[qkv_offset];
                sv[tid] = v_mem[qkv_offset];
            }
            if(bid==0 && tid == 0){
                printf("when tid=0, v cache = %f\n", sv[tid]);
            }
            O += sv[tid] * logits[iter];
            __syncthreads();
        }
        params.mha_output[qkv_offset] = O;
    }
}

void launchDecoderMaskedMHA(float* q,
                            float* k,
                            float* v,
                            float* k_cache,
                            float* v_cache,
                            float* mha_output,
                            const int batch_size,
                            const int num_heads,
                            const int head_size,
                            const int step){
    masked_MHA_kernel_params params;
    params.q = q;       
    params.k = k;       
    params.v = v;       
    params.k_cache = k_cache; 
    params.v_cache = v_cache; 
    params.batch_size = batch_size;
    params.num_heads = num_heads;
    params.head_size = head_size;
    params.scale = rsqrt(float(head_size));
    params.step = step;
    params.mha_output = mha_output;

    dim3 grid(batch_size * num_heads);
    dim3 block(head_size);
    printf("enter kernel\n");
    masked_MHA_kernel<<<grid, block, 3 * head_size * step * sizeof(float)>>>(params);
    printf("end kernel\n");
}

