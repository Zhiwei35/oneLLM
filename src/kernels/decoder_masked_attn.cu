#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include "src/kernels/decoder_masked_attn.h"

// bug1: scale's dtype must be float ,not int
// bug2: mha_kernel_params struct's pointer is on CPU, not GPU, which cause we dont run the cuda kernel, so add cudacheck is a must!
// bug3: blockreduce res should use tid=0 to write into smem
// bug4: GQA, kv_head_num brd to head_num, we can automaticly do this by head id index like lmdeploy
// half or float version: the logits and mha output both are fp32 type, q k v are all accessed vectorizedly
template<typename T>
__device__ T warpReduceSum(T val){

    for(int mask = 16; mask > 0; mask >>= 1){
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;

}
template<typename T>
__device__ T blockReduceSum(T val){
    int tid = threadIdx.x;
    int warp_id = tid / 32;
    int lane_id = tid % 32;
    int warp_nums = (blockDim.x + 31)/ 32;
    static __shared__ T warpsum[64];//why add static?or will report incomplete type
    // returned val is the sum computed by 0th thread.
    val = warpReduceSum<T>(val);
    //note: here return val of warpreducesum should be stored into smem , rather not reg, because here nums of return val  are warp nums not thread nums.
    if (lane_id == 0){
        warpsum[warp_id] = val;
    }
    __syncthreads();
    float warp_val = tid < warp_nums ? warpsum[tid] : 0;
    return warpReduceSum<T>(warp_val);

}
template<typename T>
__device__ T warpReduceMax(T val){

    for(int mask = 16; mask > 0; mask >>= 1){
        val = max(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}

template<typename T>
__device__ T blockReduceMax(T val){
    int tid = threadIdx.x;
    int warp_id = tid / 32;
    int lane_id = tid % 32;
    int warp_nums = (blockDim.x + 31)/ 32;
    static __shared__ T warpmax[64];
    // returned val is the max computed by 0th thread.
    val = warpReduceMax<T>(val);
    //note: here return val of warpreducemax should be stored into smem , rather not reg, because here nums of return val  are warp nums not thread nums.
    if (lane_id == 0){
        warpmax[warp_id] = val;
    }
    __syncthreads();
    float warp_val = tid < warp_nums ? warpmax[tid] : 0;
    return warpReduceMax<T>(warp_val);
}
// kv cache is the output of context attention(prompt phase), and the input of masked attention(token gen)
// struct masked_MHA_kernel_params
// {
//     float* q;       //[bs, q num heads, 1, head size]
//     float* k;       //[bs, kv num heads, 1, head size]
//     float* v;       //[bs, num heads, 1, head size]
//     float* k_cache; //output,[max_seq_len or step, bs, kv num heads, head size] from prompt phase
//     float* v_cache; //output,[max_seq_len or step, bs, num heads, head size] from prompt phase
//     int batch_size;
//     int num_heads;
//     int head_size;
//     float scale; // =rsqrt(head size);
//     //TODO: add qkv bias
//     int step;
//     float* mha_output;
//};


inline __device__ float2 GetRoPEfreq(int zid, int rot_embed_dim, float base, float t_step)
{
    const float inv_freq = t_step / powf(base, zid / (float)rot_embed_dim);
    return {cos(inv_freq), sin(inv_freq)};
}

// RoPE公式决定必须要做向量化
inline __device__ float2 GetRoPEres(const float2 v, const float2 coef)
{
    float2 rot_v;
    rot_v.x = coef.x * v.x - coef.y * v.y;
    rot_v.y = coef.x * v.y + coef.y * v.x;
    return rot_v;
}

inline __device__ uint32_t GetRoPEres(const uint32_t v, const float2 coef)
{
    float2 fv     = half2_to_float2(v);
    float2 rot_fv = GetRoPEres(fv, coef);
    return float2_to_half2(rot_fv);
}

inline __device__ void apply_RoPE(uint32_t& q, int tid, int rot_embed_dim, float base, float t_step)
{
    if (2 * tid >= rot_embed_dim) {
        return;
    }
    const auto coef = GetRoPEfreq(2 * tid, rot_embed_dim, base, t_step);
    q               = GetRoPEres(q, coef);
}

inline __device__ void apply_RoPE(float4& q, float4& k, int tid, int rot_embed_dim, float base, float t_step){
    if(4 * tid >= rot_embed_dim){
        return;
    }


    TwoFloat2& q_ = *reinterpret_cast<TwoFloat2*>(&q); // q为float4 寄存器
    TwoFloat2& k_ = *reinterpret_cast<TwoFloat2*>(&k);
    
    float2 coef0 = GetRoPEfreq(4 * tid, rot_embed_dim, base, t_step);
    // float freq0 = timestep / powf(rotary_embedding_base, 4 * tid / (float) rotary_embedding_dim); //分子zid = 0,2,4,, headsize/2-1,对应的theta下标为0,1,2.对应的headsize维度的索引为(0,1),(2,3)
    q_.x = GetRoPEres(q_.x ,coef0);
    // rot0.x = coef0.x * q.x -  coef0.y * q.y; //q.x为x0,q.y为x1，head size维度上两个相邻
    // rot0.y = coef0.x * q.y +  coef0.y * q.x
    float2 coef1 = GetRoPEfreq(4 * tid + 2, rot_embed_dim, base, t_step);
    q_.y = GetRoPEres(q_.y ,coef1);
    // rot1.x = coef1.x * q.x -  coef1.y * q.y; //q.x为x2,q.y为x3，head size维度上两个相邻
    // rot1.y = coef1.x * q.y +  coef1.y * q.x;
    k_.x = GetRoPEres(k_.x ,coef0);
    k_.y = GetRoPEres(k_.y ,coef1);
}

// block and thread allocation
// 1 block -> head size，后续可改进为1 warp -> 1 head size
// 1 grid -> bs * num heads
template<typename T>
__global__ void masked_MHA_kernel(const T* q,
                    const T* k,
                    const T* v,
                    T* qkv_bias,
                    T* k_cache,
                    T* v_cache,
                    float* mha_output,
                    const int batch_size,
                    const int head_num,
                    const int kv_head_num,
                    //const int num_heads,
                    const int head_size,
                    const int step,
                    T scale,
                    int   rotary_embedding_dim,
                    float rotary_embedding_base){// rsqrt(dh)
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int q_head_id = bid % head_num;
    int q_batch_id = bid / head_num;
    int kv_head_id = bid % kv_head_num;
    int kv_batch_id = bid / kv_head_num;

    int batch_stride = head_num * head_size;
    int head_stride = head_size;
    int q_offset = q_batch_id * batch_stride + q_head_id * head_stride + tid;
    int k_offset = kv_batch_id * batch_stride + kv_head_id * head_stride + tid;
    int cache_offset = batch_size * batch_stride;

    int vec_size = Vec<T>::size;
    int q_offset_vec = q_batch_id * batch_stride + q_head_id * head_stride + tid * vec_size;
    int k_offset_vec = kv_batch_id * batch_stride + kv_head_id * head_stride + tid * vec_size;

    using Vec_t = typename Vec<T>::Type;
    Vec_t qvec, kvec;
    //Vec_t scale_vec = static_cast<Vec_t>(scale);
    //reuse q k v reg from rope
    const T* q_mem = q;
    const T* k_mem = k;
    const T* v_mem = v;
    if (tid * vec_size < head_size) {
        qvec = *reinterpret_cast<Vec_t*>(const_cast<T*>(&q_mem[q_offset_vec]));
        Vec_t q_bias = *reinterpret_cast<Vec_t*>(&qkv_bias[q_head_id * head_size + tid * vec_size]);
        for(int i = 0; i < vec_size; i++) {
            reinterpret_cast<float*>(&qvec)[i] += reinterpret_cast<float*>(&q_bias)[i];
        }
        kvec = *reinterpret_cast<Vec_t*>(const_cast<T*>(&k_mem[k_offset_vec]));
        Vec_t k_bias =*reinterpret_cast<Vec_t*>(&qkv_bias[kv_head_id * head_size + tid * vec_size + head_num * head_size]);
        for(int i = 0; i < vec_size; i++) {
            reinterpret_cast<float*>(&kvec)[i] += reinterpret_cast<float*>(&k_bias)[i];
        }
        //uint32_t应该等价于half2，uint16_t应该等价于half，统一用一个就行了
        //对于half，进入rope的是uint16_t，vec_t是uint32_t/uint2
        apply_RoPE(qvec, kvec, tid, rotary_embedding_dim, rotary_embedding_base, step);
    }
    // q k smem for block reduce
    extern __shared__ T sqk[];
    T* sq = sqk;
    T* sk = sq + head_size;
    float* logits = sk + head_size;
    T* sv = logits + step;
    //sq[tid] = q_mem[qkv_offset];
    if (tid * vec_size < head_size) {
        *reinterpret_cast<Vec_t*>(&sq[tid * vec_size]) = qvec;
    }
    __syncthreads();
    // FT 2.1的写法里面，kv cache是在prompt阶段已经填充，iter=0为token gen的起始iter
    for(int iter = 0; iter < step; iter++) {
        // every iter,  q and k's shape = [1, head size]
        // reuse k cache
        // float k = k_cache[iter * cache_offset + qkv_offset];
        //或许可以在每个step省略掉前step-1的qk dot
        sk[tid]= k_cache[iter * cache_offset + k_offset];
        __syncthreads();
        // when final step, update k cache
        if (iter == step - 1 && tid * vec_size < head_size) {
            // TODO: update k cache with k with bias add
            //k_cache[iter * cache_offset + qkv_offset] = k_mem[qkv_offset];
            //sk[tid] = k_mem[qkv_offset];
            *reinterpret_cast<Vec_t*>(&k_cache[iter * cache_offset + k_offset_vec]) = kvec;
            *reinterpret_cast<Vec_t*>(&sk[tid * vec_size]) = kvec;         
        }

        // sq[tid] = q_mem[qkv_offset];
        __syncthreads();
        T qk = (tid < head_size) ? sq[tid] * sk[tid] * scale : (T)0.0f;
        //block reduce using multi warp reduce
        //TODO: maybe broadcast the attn score to each thread of the block in blockreducesum
        T attn_score = blockReduceSum<T>(qk);
        if(tid == 0) {
            logits[iter] = attn_score;
        }
        __syncthreads();
    }
    //softmax(logits), logits.shape = [bs, num heads, 1, step]
    T local_logits = tid < step ? (T)logits[tid] : 0;
    __shared__ float row_max, fenmu;
    
    T block_max = blockReduceMax<T>(local_logits);
    if (tid == 0){
        row_max = block_max;
    }
    T fenzi = tid < step ? expf(logits[tid] - row_max) : 0;
    
    T block_fenmu = blockReduceSum<T>(fenzi);
    if (tid == 0){
        fenmu = block_fenmu;
    }
    if(tid < step) {
        logits[tid] = (T)(fenzi / fenmu);
    }
    __syncthreads();

    // logits*V = [bs, num heads, 1, step] * [max_seq_len or step, bs, num heads, head size]
    if (tid < head_size) {
        // note: here is head size ,not step, because step by step, we have to use [1, step/seqlen] from logits * [1, head size] from v
        // so here we use acc O to acc the one ele logits * one ele v every step iter
        T O = 0.0f;
        for(int iter = 0; iter < step; iter++) {
            sv[tid]= v_cache[iter * cache_offset + k_offset];
            __syncthreads();
            // when final step, update k cache
            if (iter == step - 1) {
                // TODO: update k cache with k with bias add
                v_cache[iter * cache_offset + k_offset] = v_mem[k_offset];
                sv[tid] = v_mem[k_offset];
            }
            //if(bid==0 && tid == 0){
            //printf("when tid=0, v cache = %f\n", sv[tid]);

            O += sv[tid] * logits[iter];
            __syncthreads();
        }
        mha_output[q_offset] = O;
    }
}

template<> //特化以下half类型的，不在fp32代码上改
__global__ void masked_MHA_kernel(const half* q,
                    const half* k,
                    const half* v,
                    half* qkv_bias,
                    half* k_cache,
                    half* v_cache,
                    float* mha_output,
                    const int batch_size,
                    const int head_num,
                    const int kv_head_num,
                    //const int num_heads,
                    const int head_size,
                    const int step,
                    half scale,
                    int   rotary_embedding_dim,
                    float rotary_embedding_base){// rsqrt(dh)
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int q_head_id = bid % head_num;
    int q_batch_id = bid / head_num;
    int kv_head_id = bid % kv_head_num;
    int kv_batch_id = bid / kv_head_num;

    int batch_stride = head_num * head_size;
    int head_stride = head_size;
    int q_offset = q_batch_id * batch_stride + q_head_id * head_stride + tid;
    int k_offset = kv_batch_id * batch_stride + kv_head_id * head_stride + tid;
    int cache_offset = batch_size * batch_stride;

    int vec_size = Vec<half>::size;
    int q_offset_vec = q_batch_id * batch_stride + q_head_id * head_stride + tid * vec_size;
    int k_offset_vec = kv_batch_id * batch_stride + kv_head_id * head_stride + tid * vec_size;

    using Vec_t = typename Vec<half>::Type;
    Vec_t qvec, kvec;
    Vec_t scale_vec = static_cast<Vec_t>(scale);
    //reuse q k v reg from rope
    const half* q_mem = q;
    const half* k_mem = k;
    const half* v_mem = v;
    if (tid * vec_size < head_size) {
        qvec = *reinterpret_cast<Vec_t*>(const_cast<half*>(&q_mem[q_offset_vec]));
        Vec_t q_bias = *reinterpret_cast<Vec_t*>(&qkv_bias[q_head_id * head_size + tid * vec_size]);
        // for(int i = 0; i < vec_size; i++) {
        //     reinterpret_cast<float*>(&qvec)[i] += reinterpret_cast<float*>(&q_bias)[i];
        // }
        qvec = __hadd2(qvec, q_bias);
        kvec = *reinterpret_cast<Vec_t*>(const_cast<half*>(&k_mem[k_offset_vec]));
        Vec_t k_bias =*reinterpret_cast<Vec_t*>(&qkv_bias[kv_head_id * head_size + tid * vec_size + head_num * head_size]);
        // for(int i = 0; i < vec_size; i++) {
        //     reinterpret_cast<float*>(&kvec)[i] += reinterpret_cast<float*>(&k_bias)[i];
        // }
        vvec = *reinterpret_cast<Vec_t*>(const_cast<half*>(&v_mem[k_offset_vec]));
        kvec = __hadd2(kvec, k_bias);
        //uint32_t应该等价于half2，uint16_t应该等价于half，统一用一个就行了
        //对于half，进入rope的是uint16_t，vec_t是uint32_t/uint2
        apply_RoPE(qvec, kvec, tid, rotary_embedding_dim, rotary_embedding_base, step);
    }
    // q k smem for block reduce
    extern __shared__ Vec_t sqk[];
    Vec_t* sq = sqk;
    Vec_t* sk = sq + head_size / vec_size;
    float* logits = reinterpret_cast<float*>(sk + head_size / vec_size);
    Vec_t* sv = reinterpret_cast<Vec_t*>(logits + step);
    //sq[tid] = q_mem[qkv_offset];
    if (tid * vec_size < head_size) {
        // *reinterpret_cast<Vec_t*>(&sq[tid * vec_size]) = qvec;
        sq[tid] = qvec;
    }
    __syncthreads();
    // FT 2.1的写法里面，kv cache是在prompt阶段已经填充，iter=0为token gen的起始iter
    for(int iter = 0; iter < step; iter++) {
        // every iter,  q and k's shape = [1, head size]
        // reuse k cache
        // float k = k_cache[iter * cache_offset + qkv_offset];
        //或许可以在每个step省略掉前step-1的qk dot
        sk[tid]= *reinterpret_cast<Vec_t*>(&k_cache[iter * cache_offset + k_offset_vec]);
        __syncthreads();
        // when final step, update k cache
        if (iter == step - 1 && tid * vec_size < head_size) {
            // TODO: update k cache with k with bias add
            //k_cache[iter * cache_offset + qkv_offset] = k_mem[qkv_offset];
            //sk[tid] = k_mem[qkv_offset];
            *reinterpret_cast<Vec_t*>(&k_cache[iter * cache_offset + k_offset_vec]) = kvec;
            sk[tid] = kvec;         
        }

        // sq[tid] = q_mem[qkv_offset];
        __syncthreads();
        Vec_t qk = (tid * vec_size < head_size) ? __hmul2(__hmul2(sq[tid], sk[tid]), scale_vec) : static_cast<Vec_t>(0);
        //block reduce using multi warp reduce
        float qk_fp32 = __half2float(qk.x) + __half2float(qk.y);
        float attn_score = blockReduceSum<float>(qk_fp32);
        if(tid == 0) {
            logits[iter] = attn_score;
        }
        __syncthreads();
    }
    //softmax(logits), logits.shape = [bs, num heads, 1, step]
    float local_logits = tid < step ? logits[tid] : 0;
    __shared__ float row_max, fenmu;
    
    float block_max = blockReduceMax<float>(local_logits);
    if (tid == 0){
        row_max = block_max;
    }
    float fenzi = tid < step ? expf(logits[tid] - row_max) : 0;
    
    float block_fenmu = blockReduceSum<float>(fenzi);
    if (tid == 0){
        fenmu = block_fenmu;
    }
    if(tid < step) {
        logits[tid] = (float)(fenzi / fenmu);
    }
    __syncthreads();

    // logits*V = [bs, num heads, 1, step] * [max_seq_len or step, bs, num heads, head size]
    if (tid * vec_size < head_size) {
        // note: here is head size ,not step, because step by step, we have to use [1, step/seqlen] from logits * [1, head size] from v
        // so here we use acc O to acc the one ele logits * one ele v every step iter
        float2 O = static_cast<float2>(0.0f);
        //O.x = 0.0f;
        //O.y = 0.0f;
        for(int iter = 0; iter < step; iter++) {
            sv[tid]= *reinterpret_cast<Vec_t*>(&v_cache[iter * cache_offset + k_offset_vec]);
            //sv[tid]= v_cache[iter * cache_offset + k_offset];
            __syncthreads();
            // when final step, update k cache
            if (iter == step - 1) {
                // TODO: update k cache with k with bias add
                // v_cache[iter * cache_offset + k_offset] = v_mem[k_offset];
                // sv[tid] = v_mem[k_offset];
                *reinterpret_cast<Vec_t*>(&v_cache[iter * cache_offset + k_offset_vec]) = vvec;
                sv[tid] = vvec;  
            }
            //if(bid==0 && tid == 0){
            //printf("when tid=0, v cache = %f\n", sv[tid]);
            O.x += logits[iter] * sv[tid].x;
            O.y += logits[iter] * sv[tid].y;
            //O += sv[tid] * logits[iter];
            __syncthreads();
        }
        *reinterpret_cast<float2*>(&mha_output[q_offset_vec]) = O;
    }
}

// void launchDecoderMaskedMHA(float* q,
//                             float* k,
//                             float* v,
//                             float* k_cache,
//                             float* v_cache,
//                             float* mha_output,
//                             const int batch_size,
//                             const int num_heads,
//                             const int head_size,
//                             const int step){
template<typename T>
void launchDecoderMaskedMHA(TensorWrapper<T>* qkv_buf,
                            BaseWeight<T>& qkv,
                            TensorWrapper<T>* k_cache,
                            TensorWrapper<T>* v_cache,
                            TensorWrapper<bool>* finished,
                            TensorWrapper<int>* step,
                            TensorWrapper<float>* mha_output,
                            LLaMAAttentionStaticParams& static_params){
    const int batch_size = qkv_buf->shape[0];
    const int qkv_head_num = qkv_buf->shape[1];
    const int kv_head_num = k_cache->shape[2]; 
    int head_num = qkv_head_num - 2 * kv_head_num;
    const int head_size = qkv_buf->shape[2];
    const int cur_step = step->getVal<int>();
    T* qkv_data = qkv_buf->data;
    //[bs,1,qkv_head_num,head_size]
    T* q = qkv_data;
    T* k = qkv_data + head_num * head_size;
    T* v = qkv_data + (head_num + kv_head_num) * head_size;
    bool is_half = sizeof(T) == 2;
    T scale = is_half ? __float2half(rsqrt(float(head_size))) : rsqrt(float(head_size));

    int   rotary_embedding_dim = static_params.rotary_embedding_dim;
    float rotary_embedding_base = static_params.rotary_embedding_base;
    int   max_position_embeddings = static_params.max_position_embeddings;
    bool  use_dynamic_ntk = static_params.use_dynamic_ntk;
    dim3 grid(batch_size * head_num);//这里的block分配可以匹配得上lmdeploy
    dim3 block(head_size); //vec size = 4 for fp32
    printf("calling fused masked self attn kernel\n");
    masked_MHA_kernel<T><<<grid, block, (3 * head_size * sizeof(T) + cur_step * sizeof(float))>>>(
                                                                                q,
                                                                                k,
                                                                                v,
                                                                                /*(T*)*/qkv.bias,
                                                                                k_cache->data,
                                                                                v_cache->data,
                                                                                mha_output->data,
                                                                                batch_size,
                                                                                head_num,
                                                                                kv_head_num,
                                                                                //num_heads,
                                                                                head_size,
                                                                                cur_step,
                                                                                scale,
                                                                                rotary_embedding_base,
                                                                                rotary_embedding_dim);
    printf("called fused masked self attn kernel\n");
}

template void launchDecoderMaskedMHA(TensorWrapper<float>* qkv_buf,
                            BaseWeight<float>& qkv,
                            TensorWrapper<float>* k_cache,
                            TensorWrapper<float>* v_cache,
                            TensorWrapper<bool>* finished,
                            TensorWrapper<int>* step,
                            TensorWrapper<float>* mha_output,
                            LLaMAAttentionStaticParams& static_params);

template void launchDecoderMaskedMHA(TensorWrapper<half>* qkv_buf,
                            BaseWeight<half>& qkv,
                            TensorWrapper<half>* k_cache,
                            TensorWrapper<half>* v_cache,
                            TensorWrapper<bool>* finished,
                            TensorWrapper<int>* step,
                            TensorWrapper<float>* mha_output,
                            LLaMAAttentionStaticParams& static_params);