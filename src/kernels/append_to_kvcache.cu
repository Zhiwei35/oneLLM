#include "hip/hip_runtime.h"
// k/v shape = [bs, kv_head num, max_q_len, head size] // 为什么这里不是max_k_len?因为k v=w * x，此时x中seqlen维度为max_q_len
// kv cache shape = [num layers, bs, kv_head num, max_seq_len, head size] = >[bs, kv_head num, seqlen[history_len:history_len+seqlen] , head size]
// kv cache 是每个layer都有单独的kv cache ， from llama_from_ft.cc#104
// ksrc shape = [bs, kv_head num, max_q_len, head size],为什么是q_len?

#include "src/kernels/append_to_kvcache.h"
#include<iostream>
__global__ void append_key_cache(float*          k_dst, //[num layers, bs, kv head num, max_q_len, head size]
                                 const size_t layer_offset,
                                 const float*     k_src,
                                 const int    kv_head_num,
                                 const int    head_size,
                                 const int*   cur_query_length,
                                 const int*   history_length,
                                 const int    max_q_len, 
                                 const int    max_seq_len){
    int batch_id = blockIdx.y;
    int head_id = blockIdx.z;
    int tid = threadIdx.x;
    int token_id = blockIdx.x;
    
    // 当前layer的k cache
    float* k_cache_dst = k_dst + layer_offset;
    int cur_seq_len = cur_query_length[batch_id];
    int cumsum_seq_len = history_length[batch_id];
    //note: the if judge is a must, because the max_q_len is GTE than cur_seq_len.
    if(token_id < cur_seq_len){
    // [batch, head num, max_q_len, head size] -> [batch, head num, maxseqlen[cumsum_seq_len:cumsum_seq_len+cur_seq_len], head size]
        int src_offset = batch_id * kv_head_num * max_q_len * head_size + //为什么这里不是max_k_len，新进来的kv应该是max_k_len和max_v_len
                            head_id * max_q_len * head_size + 
                                token_id * head_size + tid;
        int dst_offset = batch_id * kv_head_num * max_seq_len * head_size +
                            head_id * max_seq_len * head_size + 
                                (cumsum_seq_len + token_id) * head_size + tid;
        k_dst[dst_offset] = k_src[src_offset];
    }
}

__global__ void append_value_cache(float*          v_dst,
                                    const size_t layer_offset,
                                    const float*     v_src,
                                    const int    kv_head_num,
                                    const int    head_size,
                                    const int*   cur_query_length,
                                    const int*   history_length,
                                    const int    max_q_len, 
                                    const int    max_seq_len)
{
    int batch_id = blockIdx.y;
    int head_id = blockIdx.z;
    int tid = threadIdx.x;
    int token_id = blockIdx.x;
    
    // 当前layer的v cache
    float* v_cache_dst = v_dst + layer_offset;
    int cur_seq_len = cur_query_length[batch_id];
    int cumsum_seq_len = history_length[batch_id];
    //note: the if judge is a must, because the max_q_len is GTE than cur_seq_len.
    if(token_id < cur_seq_len){
    // [batch, head num, max_q_len, head size] -> [batch, head num, maxseqlen[cumsum_seq_len:cumsum_seq_len+cur_seq_len], head size]
        int src_offset = batch_id * kv_head_num * max_q_len * head_size + 
                            head_id * max_q_len * head_size + 
                                token_id * head_size + tid;
        int dst_offset = batch_id * kv_head_num * max_seq_len * head_size +
                            head_id * max_seq_len * head_size + 
                                (cumsum_seq_len + token_id) * head_size + tid;
        v_dst[dst_offset] = v_src[src_offset];
    }
}
// k/v shape = [bs, kv_head num, max_q_len, head size] // 为什么这里不是max_k_len，新进来的kv应该是max_k_len
// kv cache shape = [bs, kv_head num, max_seq_len, head size] = >[bs, kv_head num, seqlen[history_len:history_len+seqlen] , head size]
// ksrc shape = [bs, kv_head num, max_q_len, head size],为什么是q_len?
void launchAppendKVCache(Tensor*     k_src, // from qkv bias and rope
                         Tensor*     v_src,
                         Tensor*     layer_id,// layer offset = layer_id * batchxbeam * max_seq_len * kv_head_num * head_size
                         Tensor*     cur_query_length, // current epoch or local input length,[batchsize]
                         Tensor*     history_length,
                         Tensor*     k_dst, 
                         Tensor*     v_dst)
{
    int batch_size =k_src->shape[0];    
    int max_seq_len = k_dst->shape[2];
    int kv_head_num = k_src->shape[1];
    int max_q_len = k_src->shape[2];
    int head_size = k_src->shape[3];
    int blockSize = head_size;
    size_t layer_offset = 0 * batch_size * kv_head_num * max_seq_len * head_size;
    //note: this is for vectorization of kv cache for attention
    //constexpr int x = (sizeof(T) == 4) ? 4 : 8;
    dim3 grid(max_q_len, batch_size, kv_head_num);
    std::cout << "calling concat kv cache kernel" << "\n";
    append_key_cache<<<grid, blockSize>>>((float*)k_dst->data,
                                              layer_offset,
                                              (float*)k_src->data,
                                              kv_head_num,
                                              head_size,
                                              (int*)cur_query_length->data,
                                              (int*)history_length->data,
                                              max_q_len,
                                              max_seq_len);

    append_value_cache<<<grid, blockSize>>>((float*)v_dst->data,
                                                layer_offset,
                                                (float*)v_src->data,
                                                kv_head_num,
                                                head_size,
                                                (int*)cur_query_length->data,
                                                (int*)history_length->data,
                                                max_q_len,
                                                max_seq_len);
    
    std::cout << "called concat kv cache kernel" << "\n";

}

