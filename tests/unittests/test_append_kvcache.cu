#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector

#include <hip/hip_runtime.h>
#include "src/kernels/append_to_kvcache.h"

int main() {
    const int local_batch_size = 1;
    const int max_q_len = 16;
    const int max_seq_len = 32;
    const int head_size = 8;
    const int kv_head_num = 2;
    const int kv_size = 1 * local_batch_size * max_q_len * kv_head_num * head_size;
    const int layer_offset = 1 * local_batch_size * max_seq_len * kv_head_num * head_size;
    // debug info, better to retain: std::cout <<"batch_size=" << batch_size << "  vocab_size=" << vocab_size << std::endl;
    const int kvcache_size = layer_offset;
    const int layer_id = 0;

    float* h_k_src;
    float *d_k_src;
    h_k_src = (float*)malloc(sizeof(float) * kv_size);
    hipMalloc((void**)&d_k_src, sizeof(float) * kv_size);

    float* h_v_src;
    float *d_v_src;
    h_v_src = (float*)malloc(sizeof(float) * kv_size);
    hipMalloc((void**)&d_v_src, sizeof(float) * kv_size);

    int* cur_query_length = (int*)malloc(sizeof(int) * local_batch_size);
    int* history_length = (int*)malloc(sizeof(int) * local_batch_size);
    int* dcur_query_length;
    int* dhistory_length;
    hipMalloc((void**)&dcur_query_length, sizeof(int) * local_batch_size);
    hipMalloc((void**)&dhistory_length, sizeof(int) * local_batch_size);
   
    float* h_k_dst = (float*)malloc(sizeof(float) * kvcache_size);
    float* h_v_dst = (float*)malloc(sizeof(float) * kvcache_size);
    float* d_k_dst;
    float* d_v_dst;
    hipMalloc((void**)&d_k_dst,sizeof(float) * kvcache_size);
    hipMalloc((void**)&d_v_dst,sizeof(float) * kvcache_size);
    float* kv_scale;
    hipMalloc((void**)&kv_scale, sizeof(float));
    for(int i = 0; i < kv_size; i++) {
       h_k_src[i] = 1.0f;
       h_v_src[i] = 1.0f;
    }
    for(int i = 0; i < local_batch_size; i++) {
       cur_query_length[i] = 16;
       history_length[i] = 1;
    }
    int* h_layer_id = (int*)malloc(sizeof(int)*local_batch_size);
    int* d_layer_id;
    hipMalloc((void**)&d_layer_id,sizeof(int)*local_batch_size);

    hipMemcpy(d_v_src, h_v_src, sizeof(float)*kv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_k_src, h_k_src, sizeof(float)*kv_size, hipMemcpyHostToDevice);
    hipMemcpy(dcur_query_length, cur_query_length, sizeof(int)*local_batch_size, hipMemcpyHostToDevice);
    hipMemcpy(dhistory_length, history_length, sizeof(int)*local_batch_size, hipMemcpyHostToDevice);
    hipMemcpy(d_layer_id, h_layer_id, sizeof(int)*local_batch_size, hipMemcpyHostToDevice);

    DataType type = getTensorType<float>(); 
    DataType type_int = getTensorType<int>(); 
    Tensor in_ksrc(Device::GPU, type, {local_batch_size, kv_head_num, max_q_len, head_size}, d_k_src);
    Tensor in_vsrc(Device::GPU, type, {local_batch_size, kv_head_num, max_q_len, head_size}, d_v_src);
    Tensor layer_id(Device::GPU, type_int, {local_batch_size}, d_layer_id);
    Tensor cur_q_len(Device::GPU, type_int, {local_batch_size}, dcur_query_length);
    Tensor history_len(Device::GPU, type_int, {local_batch_size}, dhistory_length);
    Tensor out_kdst(Device::GPU, type, {local_batch_size, kv_head_num, max_seq_len, head_size}, d_k_dst);
    Tensor out_vdst(Device::GPU, type, {local_batch_size, kv_head_num, max_seq_len, head_size}, d_v_dst);
    // size_t layer_offset = layer_id * local_batch_size * kv_head_num * max_seq_len * head_size;
    // debug info, better to retain: std::cout << "before launch kernel" << std::endl;
    launchAppendKVCache(&in_ksrc, &in_vsrc, &layer_id, &cur_q_len, &history_len, &out_kdst, &out_vdst);
    // debug info, better to retain: std::cout << "after launch kernel" << std::endl;
    // Note: remember to memcpy from device to host and define the correct copy size(mul the sizeof(dtype)), or will cause segment fault
    hipMemcpy(h_v_dst, d_v_dst, sizeof(float) * kvcache_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_k_dst, d_k_dst, sizeof(float) * kvcache_size, hipMemcpyDeviceToHost);
    // debug info, better to retain: std::cout << "cuda memcpy device to host" << std::endl;
    // note: need to add offset2index and index2offset API to help us program and check result
    for(int i = local_batch_size * (1) * kv_head_num * head_size; i < local_batch_size * max_seq_len * kv_head_num * head_size; i++) {
        printf("index = %d\n", i);
        printf("res k = %f\n",h_k_dst[i]);
        // debug info, better to retain: printf("topK id = %d\n", id);
        printf("res v = %f\n",h_v_dst[i]);
        printf("===============\n");
        // debug info, better to retain: printf("topK val =%f\n", val);
    }
    // debug info, better to retain: std::cout << "before free" << std::endl;
    free(h_k_src);
    free(h_v_src);
    free(h_k_dst);
    free(h_v_dst);
    free(cur_query_length);
    free(history_length);
    free(h_layer_id);
    hipFree(d_k_src);
    hipFree(d_v_src);
    hipFree(d_k_dst);
    hipFree(d_v_dst);
    hipFree(dcur_query_length);
    hipFree(dhistory_length);
    hipFree(kv_scale);
    hipFree(d_layer_id);
}
