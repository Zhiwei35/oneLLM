#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector

#include <math.h>
#include "src/kernels/transpose_kernel.h"
//[bs, kv head num, max_seq_len, head size]=>[bs, q head num, max_k_len, head size]
//context_length.shape=[bs]
int main() {
    const int batch_size = 1;
    const int head_num = 2;
    const int kv_head_num = 2;
    const int max_seq_len = 4;
    const int max_k_len = 2;
    const int head_size = 2;
    const int num_layers = 2;
//    const int layer_id = 0;
    // debug info, better to retain: std::cout <<"batch_size=" << batch_size << "  vocab_size=" << vocab_size << std::endl;
    const int k_size = num_layers * batch_size * kv_head_num * max_seq_len * head_size;
    const int out_k_size = batch_size * head_num * max_k_len * head_size;
    float* h_k;
    float* d_k;
    h_k = (float*)malloc(sizeof(float) * k_size);
    hipMalloc((void**)&d_k, sizeof(float) * k_size);
//    const int k_size = batch_size * kv_head_num * max_seq_len * head_size;
    float* h_v;
    float* d_v;
    h_v = (float*)malloc(sizeof(float) * k_size);
    hipMalloc((void**)&d_v, sizeof(float) * k_size);
    int* h_ctx_len;
    int* d_ctx_len;
    h_ctx_len = (int*)malloc(sizeof(int) * batch_size);
    hipMalloc((void**)&d_ctx_len, sizeof(int) * batch_size);
    float* h_trans_k;
    float* d_trans_k;
    h_trans_k = (float*)malloc(sizeof(float) * out_k_size);
    hipMalloc((void**)&d_trans_k, sizeof(float) * out_k_size);
    float* h_trans_v;
    float* d_trans_v;
    h_trans_v = (float*)malloc(sizeof(float) * out_k_size);
    hipMalloc((void**)&d_trans_v, sizeof(float) * out_k_size);   

    for(int i = 0; i < k_size; i++) {
       h_v[i] = i;
       h_k[i] = i;
    }
    int* h_layer_id = (int*)malloc(sizeof(int)*batch_size);
    int* d_layer_id;
    hipMalloc((void**)&d_layer_id,sizeof(int)*batch_size);

    for(int i = 0; i < batch_size; i++) {
       h_ctx_len[i] = 2;
       h_layer_id[i] = 0;
    }    
    
    hipMemcpy(d_k, h_k, sizeof(float) * k_size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, sizeof(float) * k_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ctx_len, h_ctx_len, sizeof(int) * batch_size, hipMemcpyHostToDevice);
    hipMemcpy(d_layer_id, h_layer_id, sizeof(int) * batch_size, hipMemcpyHostToDevice);
    DataType type = getTensorType<float>(); 
    DataType type_int = getTensorType<int>(); 
    Tensor in_k(Device::GPU, type, {batch_size, kv_head_num, max_seq_len, head_size}, d_k);
    Tensor in_v(Device::GPU, type, {batch_size, kv_head_num, max_seq_len, head_size}, d_v);
    Tensor ctx_len(Device::GPU, type_int, {batch_size}, d_ctx_len);
    Tensor out_k(Device::GPU, type, {batch_size, head_num, max_k_len, head_size}, d_trans_k);
    Tensor out_v(Device::GPU, type, {batch_size, head_num, max_k_len, head_size}, d_trans_v);
    Tensor layer_id(Device::GPU, type_int, {batch_size}, d_layer_id);
    
    // size_t layer_offset = layer_id * batch_size * kv_head_num * max_seq_len * head_size;
    std::cout << "before launch softmax kernel" << std::endl;
    launchTransposeKVCache(&in_k, &in_v, &ctx_len, &layer_id, &out_k, &out_v);
    std::cout << "after launch softmax kernel" << std::endl;
    std::cout << "cuda memcpy device to host" << std::endl;
    // Note: remember to memcpy from device to host and define the correct copy size(mul the sizeof(dtype)), or will cause segment fault
    hipMemcpy(h_trans_k, out_k.data, sizeof(float) * out_k_size, hipMemcpyDeviceToHost);
    for(int i = 0; i < out_k_size; i++) {
        printf("k trans[%d] = %f\n", i, h_trans_k[i]);
    }
    // debug info, better to retain: std::cout << "before free" << std::endl;
    free(h_k);
    free(h_v);
    free(h_ctx_len);
    free(h_trans_k);
    free(h_trans_v);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_ctx_len);
    hipFree(d_trans_k);
    hipFree(d_trans_v);
}
