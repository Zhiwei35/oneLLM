#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector

#include "src/kernels/decoder_masked_attn.h"

void CPUMaskedAttn(const float* q,
                    const float* k,
                    const float* v,
                    float* k_cache,
                    float* v_cache,
                    float* mha_output,
                    const int batch_size,
                    const int num_heads,
                    const int head_size,
                    const int step){
    int batch_stride = num_heads * head_size;
    int head_stride = head_size;
    int cache_offset = batch_size * batch_stride;
    int block_nums = batch_size * num_heads;
    int scale = rsqrt(float(head_size));

    const float* q_mem = q;
    const float* k_mem = k;
    const float* v_mem = v;

    // tmp buffer
    float* sqk = (float*)malloc(sizeof(float) * (block_nums * 3 * head_size * step));
    float* sq = sqk;
    float* sk = sq + block_nums * head_size;
    float* logits = sk + block_nums * head_size;
    float* sv = logits + block_nums * step;
    // FT 2.1的写法里面，kv cache是在prompt阶段已经填充，iter=0为token gen的起始iter
    for(int batch_id = 0; batch_id < batch_size; batch_id++) {
        for(int head_id = 0; head_id < num_heads; head_id++) {
            float row_max = 0.0f;
            for(int iter = 0; iter < step; iter++) {
                float attn_score = 0.0f;
                for(int tid = 0; tid < head_size; tid++) {
                    int qkv_offset = batch_id * batch_stride + head_id * head_stride + tid;
                    
                    sk[tid]= k_cache[iter * cache_offset + qkv_offset];
                    // when final step, update k cache
                    if (iter == step - 1) {
                        // TODO: update k cache with k with bias add
                        k_cache[iter * cache_offset + qkv_offset] = k_mem[qkv_offset];
                        sk[tid] = k_mem[qkv_offset];
                    }
                    
                    sq[tid] = q_mem[qkv_offset];
                    float qk = sq[tid] * sk[tid] * scale;
                    //block reduce using multi warp reduce
                    //TODO: maybe broadcast the attn score to each thread of the block in blockreducesum
                    attn_score += qk;
                }
                printf("every step/seqlen attn score = %f\n", attn_score);
                logits[batch_id * num_heads * step + head_id * step + iter * head_size] = attn_score;
                //softmax(logits), logits.shape = [bs, num heads, 1, step] 
                row_max = std::max(attn_score, row_max);
            }
            printf("all step/seqlen(one row) max attn score = %f\n", row_max);
            float fenzi = 0.0f;
            float fenmu = 0.0f;
            for(int iter = 0; iter < step; iter++) { // row
                fenzi = expf(logits[batch_id * num_heads * step + head_id * step + iter * head_size] - row_max);
                fenmu += fenzi;
            }
            for(int iter = 0; iter < step; iter++) { // row
                logits[batch_id * num_heads * step + head_id * step + iter * head_size] = fenzi / fenmu;
            }
            // logits*V = [bs, num heads, 1, step] * [mx_seq_len or step, bs, num heads, head size]
            for(int tid = 0; tid < head_size; tid++) {
                float O = 0.0f;
                int qkv_offset = batch_id * batch_stride + head_id * head_stride + tid;
                for(int iter = 0; iter < step; iter++) {
                    sv[tid]= v_cache[iter * cache_offset + qkv_offset];
                    // when final step, update k cache
                    if (iter == step - 1) {
                        // TODO: update k cache with k with bias add
                        v_cache[iter * cache_offset + qkv_offset] = v_mem[qkv_offset];
                        sv[tid] = v_mem[qkv_offset];
                    }
                    O += sv[tid] * logits[iter];
                }
                mha_output[qkv_offset] = O;
            }
        }
    }

    free(sqk);
}

bool CheckResult(float* CPUoutput, float* GPUoutput, int output_size) {
    for(int i = 0; i < output_size; i++) {
        if(fabs(CPUoutput[i] - GPUoutput[i]) > 1e-6){
        
            printf("the %dth res is wrong, CPUoutput = %f, GPUoutput = %f\n", i, CPUoutput[i], GPUoutput[i]);
            //return false;
        }

    }
    return true;
}
int main() {
    constexpr int batch_size = 1;
    constexpr int head_size = 16;
    constexpr int num_heads = 2;
    constexpr int step = 4;
    constexpr int max_seq_len = 32;
    float* h_q;
    float* d_q;
    int q_size = batch_size * num_heads * head_size;
    h_q = (float*)malloc(sizeof(float) * q_size);
    hipMalloc((void**)&d_q, sizeof(float) * q_size);

    float* h_k;
    float* d_k;
    int k_size = batch_size * num_heads * head_size;
    h_k = (float*)malloc(sizeof(float) * k_size);
    hipMalloc((void**)&d_k, sizeof(float) * k_size);

    float* h_v;
    float* d_v;
    int v_size = batch_size * num_heads * head_size;
    h_v = (float*)malloc(sizeof(float) * v_size);
    hipMalloc((void**)&d_v, sizeof(float) * v_size);  

    float* h_kcache;
    float* d_kcache;
    int kcache_size = max_seq_len * batch_size * num_heads * head_size;
    h_kcache = (float*)malloc(sizeof(float) * kcache_size);
    hipMalloc((void**)&d_kcache, sizeof(float) * kcache_size);  

    float* h_vcache;
    float* d_vcache;
    int vcache_size = max_seq_len * batch_size * num_heads * head_size;
    h_vcache = (float*)malloc(sizeof(float) * vcache_size);
    hipMalloc((void**)&d_vcache, sizeof(float) * vcache_size);  

    for(int i = 0; i < q_size; i++) { // initialize host data
        h_q[i] = 1.0f;
        h_k[i] = 1.0f;
        h_v[i] = 1.0f;
    }
    // note: prompt phase only generate part of k v cache
    for(int i = 0; i < (kcache_size * (step - 1)) / max_seq_len; i++) { // initialize host data
        h_kcache[i] = 1.0f;
        h_vcache[i] = 1.0f;
    }

    float* h_o;
    float* d_o;
    int o_size = batch_size * num_heads * head_size;
    h_o = (float*)malloc(sizeof(float) * o_size);
    hipMalloc((void**)&d_o, sizeof(float) * o_size); 

    hipMemcpy(d_q, h_q, sizeof(float) * q_size, hipMemcpyHostToDevice);
    hipMemcpy(d_k, h_k, sizeof(float) * k_size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, sizeof(float) * v_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kcache, h_kcache, sizeof(float) * kcache_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vcache, h_vcache, sizeof(float) * vcache_size, hipMemcpyHostToDevice);
  
    launchDecoderMaskedMHA(d_q, d_k, d_v, d_kcache, d_vcache, d_o, batch_size, num_heads, head_size, step);
    hipMemcpy(h_o, d_o, sizeof(float) * o_size, hipMemcpyDeviceToHost);
    float* CPU_output = (float*)malloc(sizeof(float) * o_size);
    CPUMaskedAttn(h_q, h_k, h_v, h_kcache, h_vcache, h_o, batch_size, num_heads, head_size, step);
    bool is_true = CheckResult(CPU_output, h_o, o_size);
    if(is_true){
        printf("test passed");
    } else {
        printf("test failed");
    }

    free(h_q);
    free(h_k);
    free(h_v);
    free(h_kcache);
    free(h_vcache);
    free(h_o);
    free(CPU_output);
    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_o);
    hipFree(d_kcache);
    hipFree(d_vcache);
}
