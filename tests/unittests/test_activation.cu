#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector

#include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>
#include "src/kernels/activation_kernel.h"
template<typename T>
void CPUSwiGLU(T* input, T* output, int batch_size, int intermedia_size){
    float silu_out = 0.0f;
    for(int batch_id = 0; batch_id < batch_size; batch_id++){
        for(int i = 0; i < intermedia_size; i++) {
            int offset1 = batch_id * 2 * intermedia_size + i;
            int offset2 = batch_id * 2 * intermedia_size + i + intermedia_size;
            int out_offset = batch_id * intermedia_size + i;
            silu_out = (float)input[offset1] / (1.0f + expf(-1 * (float)input[offset1]));
            output[out_offset] = static_cast<T>(silu_out * (float)input[offset2]);
        }
    }
}
template<typename T>
bool CheckResult(T* CPUoutput, T* GPUoutput, int output_size) {
    for(int i = 0; i < output_size; i++) {
        if(fabs((float)CPUoutput[i] - (float)GPUoutput[i]) > 1e-6){
            printf("the %dth res is wrong, CPUoutput = %f, GPUoutput = %f\n", i, CPUoutput[i], GPUoutput[i]);
            return false;
        }

    }
    return true;
}

template<typename T>
void test_act(int batch_size, int intermedia_size, int input_size , int output_size) {
    T* h_input;
    T* d_input;
    h_input = (T*)malloc(sizeof(T) * input_size);
    hipMalloc((void**)&d_input, sizeof(T) * input_size);
    T* h_output;
    T* d_output;
    h_output = (T*)malloc(sizeof(T) * output_size);
    hipMalloc((void**)&d_output, sizeof(T) * output_size);
    for(int i = 0; i < input_size; i++) { // initialize host data
        h_input[i] = (T)i;
    }
    hipMemcpy(d_input, h_input, sizeof(T) * input_size, hipMemcpyHostToDevice);
    launchAct(d_input, d_output, batch_size, intermedia_size);
    hipMemcpy(h_output, d_output, sizeof(T) * output_size, hipMemcpyDeviceToHost);
    T* CPU_output = (T*)malloc(sizeof(T) * output_size);
    CPUSwiGLU(h_input, CPU_output, batch_size, intermedia_size);
    bool is_true = CheckResult(CPU_output, h_output, output_size);
    if(is_true){
        printf("test passed");
    } else {
        printf("test failed");
    }

    free(h_input);
    free(h_output);
    free(CPU_output);
    hipFree(d_input);
    hipFree(d_output);    
}
int main(int argc, char** argv) {
    constexpr int batch_size = 16;
    constexpr int intermedia_size = 11008;
    constexpr int input_size = batch_size * intermedia_size * 2;
    constexpr int output_size = batch_size * intermedia_size;
    if (atoi(argv[0]) == 0){
        test_act<float>(batch_size, intermedia_size, input_size, output_size);
    }
    else if (atoi(argv[0]) == 1) {
        test_act<half>(batch_size, intermedia_size, input_size, output_size);
    }
    
}
