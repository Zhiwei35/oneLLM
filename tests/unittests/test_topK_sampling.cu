#include <iostream>
#include "src/kernels/topK_sampling.h"
#include "src/utils/macro.h"

int main() {
    const int batch_size = 3;
    const int K = 3;
    int vocab_size = 1000;
    int step = 5;
    int end_id = 10;
    int* h_topkid;
    int* d_topkid;
    h_topkid = (int*)malloc(sizeof(int) * batch_size * K);
    hipMalloc((void**)&d_topkid, sizeof(int) * batch_size * K);

    float* h_topkval;
    float* d_topkval;
    h_topkval = (float*)malloc(sizeof(float) * batch_size * K);
    hipMalloc((void**)&d_topkval, sizeof(float) * batch_size * K);   

    int* h_outid;
    int* d_outid;
    h_outid = (int*)malloc(sizeof(int) * batch_size);
    hipMalloc((void**)&d_outid, sizeof(int) * batch_size); 

    int* h_cuseqlen;
    int* d_cuseqlen;
    h_cuseqlen = (int*)malloc(sizeof(int) * batch_size);
    hipMalloc((void**)&d_cuseqlen, sizeof(int) * batch_size); 

    bool* h_finished;
    bool* d_finished;
    h_finished = (bool*)malloc(sizeof(bool) * batch_size);
    hipMalloc((void**)&d_finished, sizeof(bool) * batch_size); 

    for(int i = 0; i < batch_size; i++) {
        h_finished[i] = 0;
        h_cuseqlen[i] = 4;
    }
    for(int i = 0; i < batch_size * K; i++) {
        h_topkid[i] = i;
        h_topkval[i] = (float)(K - 1 - (i % K));// K = 5, 1st row=43210
    }

    CHECK(hipMemcpy(d_topkid, h_topkid, sizeof(int) * batch_size * K, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_topkval, h_topkval, sizeof(float) * batch_size * K, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_cuseqlen, h_cuseqlen, sizeof(int) * batch_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_finished, h_finished, sizeof(bool) * batch_size, hipMemcpyHostToDevice));
    DataType type = getTensorType<float>(); 
    DataType type_int = getTensorType<int>(); 
    DataType type_bool = getTensorType<bool>(); 
    Tensor topk_id(Device::GPU, type_int, {batch_size, K}, d_topkid);
    Tensor topk_val(Device::GPU, type, {batch_size, K}, d_topkval);
    Tensor cuseqlen(Device::GPU, type_int, {batch_size}, d_cuseqlen);
    Tensor finished(Device::GPU, type_bool, {batch_size}, d_finished);
    Tensor output_id(Device::GPU, type_int, {batch_size}, d_outid);
    IntDict intParams;
    intParams.insert({"step", step});
    intParams.insert({"vocab_size", vocab_size});
    intParams.insert({"end_id", end_id});
    std::cout << "before launch sampling kernel" << std::endl;
    launchSampling(&topk_id, &topk_val, &cuseqlen, &finished, &output_id, intParams);
    std::cout << "after launch sampling kernel" << std::endl;
    std::cout << "cuda memcpy device to host" << std::endl;
    // Note: remember to memcpy from device to host and define the correct copy size(mul the sizeof(dtype)), or will cause segment fault
    CHECK(hipMemcpy(h_outid, output_id.data, sizeof(int) * batch_size, hipMemcpyDeviceToHost));
    for (int i = 0; i < batch_size; i++) {
        std::cout << "seq" << i + 1 << ":" << h_outid[i] << std::endl;
    }
    free(h_topkid);
    free(h_topkval);
    free(h_finished);
    free(h_cuseqlen);
    free(h_outid);
    hipFree(d_topkid);
    hipFree(d_topkval);
    hipFree(d_finished);
    hipFree(d_cuseqlen);
    hipFree(d_outid);
}
